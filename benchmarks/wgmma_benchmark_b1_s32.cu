#include "hip/hip_runtime.h"
#include <iostream>
#include <functional>
#include <random>
#include <limits>
#include <cmath>

#include <cudawrappers/cu.hpp>

#include "wgmma.hpp"

__global__ void kernel_ref(const int *A, const int *B, int *C, const size_t M, const size_t N, const size_t K, const size_t multiplier) {
    size_t m = threadIdx.x + blockIdx.x * blockDim.x;
    size_t n = threadIdx.y + blockIdx.y * blockDim.y;
    if (m >= M | n >= N) {
        return;
    }
    const size_t packing_factor = 32;
    const size_t K_packed = K / packing_factor;

    int sum = 0;
    for (int k = 0; k < K_packed; k++) {
      sum += __popc(A[m * K_packed + k] & B[n * K_packed + k]);
    }
    C[m * N + n] = sum * multiplier;
}

template<size_t M, size_t N, size_t K, size_t REPEAT_COUNT=1, size_t WGMMA_COUNT=1>
__global__ void kernel_wgmma(const int *A, const int *B, int *C) {
    const size_t nthreads = blockDim.x * blockDim.y * blockDim.z;
    const size_t tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;

    wgmma::fragment<wgmma::matrix_a, M, N, K, wgmma::precision::b1, wgmma::row_major> a;
    __shared__ __align__(16) wgmma::fragment<wgmma::matrix_b, M, N, K, wgmma::precision::b1, wgmma::col_major> b;
    wgmma::fragment<wgmma::accumulator, M, N, K, int> c;
    wgmma::fill_fragment(c, 0);

    wgmma::SwizzleMode swizzle = wgmma::SwizzleMode::Interleaved;

    wgmma::load_matrix(a, A, K);
    wgmma::load_matrix(b, B, K, swizzle, tid, nthreads);
    __syncthreads();
    wgmma::smem_fence();

    unsigned long descB = wgmma::make_descriptor(b, swizzle);

    for (size_t repeat=0; repeat < REPEAT_COUNT; repeat++) {
        wgmma::arrive();
        for (size_t counter = 0; counter < WGMMA_COUNT; counter++) {
            wgmma::mma_async(a, descB, c);
        }
        wgmma::commit();
        wgmma::wait();
    }

    wgmma::store_matrix(C, c, N, wgmma::mem_row_major);
}


int main() {
    constexpr unsigned M = 64;
    constexpr unsigned K = 256;
    constexpr unsigned REPEAT_COUNT = 256;
    constexpr unsigned WGMMA_COUNT = 16;
    constexpr unsigned ITERATIONS = 4;

    constexpr std::array<unsigned, 4> N_values{8, 64, 128, 256};
    const unsigned maxN = *std::max_element(N_values.begin(), N_values.end());

    cu::init();
    cu::Device device(0);
    cu::Context context(hipDeviceScheduleBlockingSync, device);
    cu::Stream stream;

    auto generator = std::bind(std::uniform_int_distribution<int>(INT_MIN, INT_MAX),
                               std::default_random_engine());

    const size_t packing_factor = 32;
    const size_t K_packed = K / packing_factor;
    const size_t bytes_a = sizeof(int) * M * K_packed;
    const size_t bytes_b = sizeof(int) * maxN * K_packed;
    const size_t bytes_c = sizeof(int) * M * maxN;

    int *a, *b;
    int *c, *c_ref, *c_ref_host;
    hipHostMalloc(&a, bytes_a);
    hipHostMalloc(&b, bytes_b);
    hipHostMalloc(&c, bytes_c);
    hipHostMalloc(&c_ref, bytes_c);
    hipHostMalloc(&c_ref_host, bytes_c);

    int *d_a, *d_b;
    int *d_c;
    hipMalloc(&d_a, bytes_a);
    hipMalloc(&d_b, bytes_b);
    hipMalloc(&d_c, bytes_c);

    for (size_t i = 0; i < M * K_packed; i++) {
        a[i] = generator();
    }

    for (size_t i = 0; i < maxN * K_packed; i++) {
        b[i] = generator();
    }

    std::cout << "Performance is average of " << ITERATIONS << " iterations." << std::endl;
    for (const unsigned &N : N_values) {
        std::cout << "MxNxK = " << M << "x" << N << "x" << K << std::endl;
        dim3 threads_ref{32, 32, 1};
        dim3 grid_ref{M / threads_ref.x + 1, N / threads_ref.y + 1, 1};

        hipMemcpy(d_a, a, bytes_a, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, bytes_b, hipMemcpyHostToDevice);
        kernel_ref<<<grid_ref, threads_ref>>>(d_a, d_b, d_c, M, N, K, REPEAT_COUNT * WGMMA_COUNT);
        hipDeviceSynchronize();
        hipMemcpy(c_ref, d_c, bytes_c, hipMemcpyDeviceToHost);

        dim3 threads{128, 1, 1};
        dim3 grid{1, 1, 1};
        hipMemset(d_c, 0, bytes_c);
        switch(N) {
            case 8:
                kernel_wgmma<M,   8, K, REPEAT_COUNT, WGMMA_COUNT><<<grid, threads>>>(d_a, d_b, d_c);
                break;
            case 64:
                kernel_wgmma<M,  64, K, REPEAT_COUNT, WGMMA_COUNT><<<grid, threads>>>(d_a, d_b, d_c);
                break;
            case 128:
                kernel_wgmma<M, 128, K, REPEAT_COUNT, WGMMA_COUNT><<<grid, threads>>>(d_a, d_b, d_c);
                break;
            case 256:
                kernel_wgmma<M, 256, K, REPEAT_COUNT, WGMMA_COUNT><<<grid, threads>>>(d_a, d_b, d_c);
                break;
        }
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, bytes_c, hipMemcpyDeviceToHost);

        int errs = 0;
        for (size_t m=0; m < M; m++) {
            for (size_t n=0; n < N; n++) {
                float diff = c[m * N + n] - c_ref[m * N + n];
                if (diff != 0) errs++;
            }
        }
        std::cout << "Result " << (errs > 0 ? "Not " : "") << "OK" << std::endl;

        // benchmark
        int multiProcessorCount = device.getAttribute(hipDeviceAttributeMultiprocessorCount);

        // Kernel dimensions
        int nr_thread_blocks = multiProcessorCount * 512;
        dim3 grid_bench(nr_thread_blocks);
        dim3 threads_bench(wgmma::WARPGROUP_SIZE);
        double gops = 1e-9 * 2 * M * N * K * WGMMA_COUNT * REPEAT_COUNT * nr_thread_blocks;
        std::array<double, ITERATIONS> tops;
        cu::Event start, end;
        for (size_t i=0; i < ITERATIONS; i++) {
            stream.record(start);
            switch(N) {
                case 8:
                    kernel_wgmma<M,   8, K, REPEAT_COUNT, WGMMA_COUNT><<<grid_bench, threads_bench, 0, stream>>>(d_a, d_b, d_c);
                    break;
                case 64:
                    kernel_wgmma<M,  64, K, REPEAT_COUNT, WGMMA_COUNT><<<grid_bench, threads_bench, 0, stream>>>(d_a, d_b, d_c);
                    break;
                case 128:
                    kernel_wgmma<M, 128, K, REPEAT_COUNT, WGMMA_COUNT><<<grid_bench, threads_bench, 0, stream>>>(d_a, d_b, d_c);
                    break;
                case 256:
                    kernel_wgmma<M, 256, K, REPEAT_COUNT, WGMMA_COUNT><<<grid_bench, threads_bench, 0, stream>>>(d_a, d_b, d_c);
                    break;
            }
            stream.record(end);
            end.synchronize();
            stream.synchronize();
            float time = end.elapsedTime(start);
            tops[i] = gops / time; // time in ms converts giga to tera
        }
        double tops_avg = 0;
        double tops_sq = 0;
        for (auto & item : tops) {
            tops_avg += item;
            tops_sq += item * item;
        }
        tops_avg /= ITERATIONS;
        tops_sq /= ITERATIONS;
        // stddev = sqrt(mean of sq - sq of mean)
        double tops_stddev = std::sqrt(tops_sq - tops_avg * tops_avg);
        std::cout << "TOPS: " << tops_avg << " +/- " << tops_stddev << std::endl << std::endl;
    }

    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
    hipFree(d_a);
    hipFree(d_b);
}
