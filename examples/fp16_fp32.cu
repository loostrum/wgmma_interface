#include "hip/hip_runtime.h"
#include <iostream>
#include <functional>
#include <random>

#include <cudawrappers/cu.hpp>
#include <hip/hip_fp16.h>

#include "wgmma.h"

__global__ void kernel_ref(const half *A, const half *B, float *C, const size_t M, const size_t N, const size_t K, const size_t multiplier) {
    size_t m = threadIdx.x + blockIdx.x * blockDim.x;
    size_t n = threadIdx.y + blockIdx.y * blockDim.y;
    if (m >= M | n >= N) {
        return;
    }

    float sum = 0;
    for (int k=0; k < K; k++) {
      sum += static_cast<float>(A[m * K + k]) * static_cast<float>(B[n * K + k]);
    }
    C[m * N + n] = sum * multiplier;
}

template<size_t M, size_t N, size_t K, size_t REPEAT_COUNT=1, size_t WGMMA_COUNT=1>
__global__ void kernel_wgmma(const half *A, const half *B, float *C) {
    const size_t nthreads = blockDim.x * blockDim.y * blockDim.z;
    const size_t tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;

    int a[4];
    wgmma::load_matrix_a(a, A, K);

    const size_t numB = N * K;
    __shared__ __align__(16) half b[numB];
    for (size_t idx=tid; idx < numB; idx += nthreads) {
        const size_t core_matrix_N = 8;
        const size_t core_matrix_K = 8;
        // no swizzle means core matrices have to have adjacent elements
        // like the ccglib transpose kernel: tiles are contiguous in memory
        // B matrix is N x K, K major i.e. contiguous in K
        size_t n = idx / K;
        size_t k = idx % K;
        // calculate output index. First get index of core matrix
        size_t core_matrix_n = n / core_matrix_N;
        size_t core_matrix_k = k / core_matrix_K;
        size_t core_matrix_index = core_matrix_k * (N / core_matrix_N) + core_matrix_n;  // n-major!
        size_t core_matrix_start = core_matrix_index * core_matrix_N * core_matrix_K; // start position of this core matrix
        size_t core_n = n % core_matrix_N;
        size_t core_k = k % core_matrix_K;
        size_t out_idx = core_matrix_start + core_n * core_matrix_K + core_k;
        b[out_idx] = B[idx];
    }
    __syncthreads();

    float c[N/2] = {0};

    unsigned lds = 2 * N * 8; // 2048
    unsigned sds = 128;
    wgmma::SwizzleMode swizzle = wgmma::SwizzleMode::Interleaved;
    unsigned base_offset = 0;
    unsigned long addr = reinterpret_cast<unsigned long>(&b[0]);

    unsigned long descB = wgmma::make_descriptor(addr, lds, sds, base_offset, swizzle);

    for (size_t repeat=0; repeat < REPEAT_COUNT; repeat++) {
        wgmma::arrive();
        for (size_t counter = 0; counter < WGMMA_COUNT; counter++) {
            wgmma::wgmma_async(a, descB, c);
        }
        wgmma::commit();
        wgmma::wait<0>();
    }

    wgmma::store_matrix(c, C, N);
}


int main() {
    constexpr unsigned M = 64;
    constexpr unsigned N = 128;
    constexpr unsigned K = 16;
    constexpr unsigned REPEAT_COUNT = 256;
    constexpr unsigned WGMMA_COUNT = 16;
    constexpr unsigned ITERATIONS = 16;

    cu::init();
    cu::Device device(0);
    cu::Context context(hipDeviceScheduleBlockingSync, device);
    cu::Stream stream;

    auto generator = std::bind(std::uniform_int_distribution<int>(-10, 10),
                               std::default_random_engine());

    size_t bytes_a = sizeof(half) * M * K;
    size_t bytes_b = sizeof(half) * N * K;
    size_t bytes_c = sizeof(float) * M * N;

    half *a, *b; 
    float *c, *c_ref, *c_ref_host;
    hipHostMalloc(&a, bytes_a);
    hipHostMalloc(&b, bytes_b);
    hipHostMalloc(&c, bytes_c);
    hipHostMalloc(&c_ref, bytes_c);
    hipHostMalloc(&c_ref_host, bytes_c);

    half *d_a, *d_b;
    float *d_c;
    hipMalloc(&d_a, bytes_a);
    hipMalloc(&d_b, bytes_b);
    hipMalloc(&d_c, bytes_c);

    for (size_t i = 0; i < M * K; i++) {
        a[i] = (half)generator();
    }   

    for (size_t i = 0; i < N * K; i++) {
        b[i] = (half)generator();
    }   

    dim3 threads_ref{32, 32, 1}; 
    dim3 grid_ref{M / threads_ref.x + 1, N / threads_ref.y + 1, 1}; 

    hipMemcpy(d_a, a, bytes_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, bytes_b, hipMemcpyHostToDevice);
    kernel_ref<<<grid_ref, threads_ref>>>(d_a, d_b, d_c, M, N, K, REPEAT_COUNT * WGMMA_COUNT);
    hipDeviceSynchronize();
    hipMemcpy(c_ref, d_c, bytes_c, hipMemcpyDeviceToHost);

    dim3 threads{128, 1, 1}; 
    dim3 grid{1, 1, 1}; 
    hipMemset(d_c, 0, bytes_c);
    kernel_wgmma<M, N, K, REPEAT_COUNT, WGMMA_COUNT><<<grid, threads>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, bytes_c, hipMemcpyDeviceToHost);

    int errs = 0;
    for (size_t m=0; m < M; m++) {
        for (size_t n=0; n < N; n++) {
            float diff = c[m * N + n] - c_ref[m * N + n];
            if (diff != 0) errs++;
        }
    }
    std::cout << "Result " << (errs > 0 ? "Not " : "") << "OK" << std::endl;

    // benchmark
    int multiProcessorCount = device.getAttribute(hipDeviceAttributeMultiprocessorCount);

    // Kernel dimensions
    int nr_thread_blocks = multiProcessorCount * 512;
    dim3 grid_bench(nr_thread_blocks);
    dim3 threads_bench(128);
    double gops = 1e-9 * 2 * M * N * K * WGMMA_COUNT * REPEAT_COUNT * nr_thread_blocks;
    std::array<double, ITERATIONS> tflops;
    cu::Event start, end;
    for (size_t i=0; i < ITERATIONS; i++) {
        stream.record(start);
        kernel_wgmma<M, N, K, REPEAT_COUNT, WGMMA_COUNT><<<grid_bench, threads_bench, 0, stream>>>(d_a, d_b, d_c);
        stream.record(end);
        end.synchronize();
        stream.synchronize();
        float time = end.elapsedTime(start);
        double perf = gops / time; // time in ms converts giga to tera
        tflops[i] = perf;
        std::cout << "TFLOPS: " << perf << std::endl;
    } 
    double tflops_avg = 0;
    double tflops_sq = 0;
    for (auto & item : tflops) {
        tflops_avg += item;
        tflops_sq += item * item;
    }
    tflops_avg /= ITERATIONS;
    tflops_sq /= ITERATIONS;
    // stddev = mean of sq - sq of mean
    double tflops_stddev = tflops_sq - tflops_avg * tflops_avg;
    std::cout << "Average TFLOPS: " << tflops_avg << " +/- " << tflops_stddev << std::endl;

    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
    hipFree(d_a);
    hipFree(d_b);
}

