#include "hip/hip_runtime.h"
#include <iostream>
#include <functional>
#include <random>

#include <cudawrappers/cu.hpp>
#include <hip/hip_fp16.h>

#include "wgmma.h"

__global__ void kernel_ref(const half *A, const half *B, float *C, const size_t M, const size_t N, const size_t K) {
    size_t m = threadIdx.x + blockIdx.x * blockDim.x;
    size_t n = threadIdx.y + blockIdx.y * blockDim.y;
    if (m >= M | n >= N) {
        return;
    }

    float sum = 0;
    for (int k=0; k < K; k++) {
      sum += static_cast<float>(A[m * K + k]) * static_cast<float>(B[n * K + k]);
    }
    C[m * N + n] = sum;
}

template<size_t M_PER_BLOCK, size_t N_PER_BLOCK,
          size_t M_PER_WG, size_t N_PER_WG,
          size_t M_WGMMA, size_t N_WGMMA, size_t K_WGMMA>
__global__ void kernel_wgmma(const half *A, const half *B, float *C) {
    const size_t nthreads = blockDim.x * blockDim.y * blockDim.z;
    const size_t tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;

    wgmma::fragment<wgmma::matrix_a, M_WGMMA, N_WGMMA, K_WGMMA, half, wgmma::row_major> a;
    __shared__ __align__(16) wgmma::fragment<wgmma::matrix_b, M_WGMMA, N_WGMMA, K_WGMMA, half, wgmma::col_major> b;
    wgmma::fragment<wgmma::accumulator, M_WGMMA, N_WGMMA, K_WGMMA, float> c;
    wgmma::fill_fragment(c, 0);

    wgmma::SwizzleMode swizzle = wgmma::SwizzleMode::Interleaved;

    wgmma::load_matrix(a, A, K_WGMMA);
    wgmma::load_matrix(b, B, K_WGMMA, swizzle, tid, nthreads);
    __syncthreads();
    wgmma::smem_fence();

    unsigned long descB = wgmma::make_descriptor(b, swizzle);
    wgmma::arrive();
    wgmma::mma_async(a, descB, c);
    wgmma::commit();
    wgmma::wait();
    wgmma::store_matrix(c, C, N_WGMMA, wgmma::mem_row_major);
}


int main() {
    //constexpr unsigned M = 256;
    //constexpr unsigned N = 256;
    //constexpr unsigned K = 256;
    constexpr unsigned M = 64;
    constexpr unsigned N = 128;
    constexpr unsigned K = 16;

    constexpr unsigned M_PER_BLOCK = M;
    constexpr unsigned N_PER_BLOCK = N;

    constexpr unsigned M_PER_WG = M;
    constexpr unsigned N_PER_WG = N;

    constexpr unsigned M_WGMMA = 64;
    constexpr unsigned N_WGMMA = 128;
    constexpr unsigned K_WGMMA = 16;

    cu::init();
    cu::Device device(0);
    cu::Context context(hipDeviceScheduleBlockingSync, device);
    cu::Stream stream;

    auto generator = std::bind(std::uniform_int_distribution<int>(-10, 10),
                               std::default_random_engine());

    size_t bytes_a = sizeof(half) * M * K;
    size_t bytes_b = sizeof(half) * N * K;
    size_t bytes_c = sizeof(float) * M * N;

    half *a, *b;
    float *c, *c_ref, *c_ref_host;
    hipHostMalloc(&a, bytes_a);
    hipHostMalloc(&b, bytes_b);
    hipHostMalloc(&c, bytes_c);
    hipHostMalloc(&c_ref, bytes_c);
    hipHostMalloc(&c_ref_host, bytes_c);

    half *d_a, *d_b;
    float *d_c;
    hipMalloc(&d_a, bytes_a);
    hipMalloc(&d_b, bytes_b);
    hipMalloc(&d_c, bytes_c);

    for (size_t i = 0; i < M * K; i++) {
        a[i] = (half)generator();
    }

    for (size_t i = 0; i < N * K; i++) {
        b[i] = (half)generator();
    }

    dim3 threads_ref{32, 32, 1};
    dim3 grid_ref{M / threads_ref.x + 1, N / threads_ref.y + 1, 1};

    hipMemcpy(d_a, a, bytes_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, bytes_b, hipMemcpyHostToDevice);
    kernel_ref<<<grid_ref, threads_ref>>>(d_a, d_b, d_c, M, N, K);
    hipDeviceSynchronize();
    hipMemcpy(c_ref, d_c, bytes_c, hipMemcpyDeviceToHost);

    dim3 threads{128, N_PER_BLOCK/N_PER_WG, M_PER_BLOCK/M_PER_WG};
    dim3 grid{N/N_PER_BLOCK, M/M_PER_BLOCK, 1};
    hipMemset(d_c, 0, bytes_c);
    kernel_wgmma<M_PER_BLOCK, N_PER_BLOCK, M_PER_WG, N_PER_WG, M_WGMMA, N_WGMMA, K_WGMMA><<<grid, threads>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, bytes_c, hipMemcpyDeviceToHost);

    int errs = 0;
    for (size_t m=0; m < M; m++) {
        for (size_t n=0; n < N; n++) {
            float diff = c[m * N + n] - c_ref[m * N + n];
            if (diff != 0) errs++;
        }
    }
    std::cout << "Result " << (errs > 0 ? "Not " : "") << "OK" << std::endl;

    // benchmark
    int multiProcessorCount = device.getAttribute(hipDeviceAttributeMultiprocessorCount);

    // Kernel dimensions
    int nr_thread_blocks = multiProcessorCount * 512;
    dim3 grid_bench(nr_thread_blocks);
    dim3 threads_bench(128);
    double gops = 1e-9 * 2 * M * N * K * nr_thread_blocks;
    cu::Event start, end;
    stream.record(start);
    kernel_wgmma<M_PER_BLOCK, N_PER_BLOCK, M_PER_WG, N_PER_WG, M_WGMMA, N_WGMMA, K_WGMMA><<<grid_bench, threads_bench, 0, stream>>>(d_a, d_b, d_c);
    stream.record(end);
    end.synchronize();
    stream.synchronize();
    float time = end.elapsedTime(start);
    double tflops = gops / time; // time in ms converts giga to tera
    std::cout << "TFLOPS: " << tflops << std::endl;

    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
    hipFree(d_a);
    hipFree(d_b);
}
